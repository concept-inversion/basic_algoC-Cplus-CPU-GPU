#include "hip/hip_runtime.h"
/*
Vector addition with a single thread for each addition
*/


__global__ void
 simple_addition(int *a, int *b,int *c,int len)
{
    int tid=threadIdx.x +blockIdx.x*blockDim.x ;
    //while (tid<len)
    c[tid]=a[tid]+b[tid];  
    //printf("I am block: %d with tid: %d Result: %d \n",blockIdx.x,tid,c[tid]);  
    
}

/*
Vector addition with thread mapping and thread accessing its neighbor parallely
*/

//slower than simpler
__global__ void
good_addition(int *a, int *b, int *c, int len)
{
 int tid= threadIdx.x + blockIdx.x * blockDim.x; 
 const int thread_count= blockDim.x*gridDim.x;
 int step = len/thread_count;

 int start_index = tid*step;
 int end_index= (tid+1)* step;
 if (tid==thread_count-1) end_index=len;
 //printf("Step is %d\n",step);
 while(start_index< end_index)
    {
    c[start_index]=a[start_index]+b[start_index];

    //printf("I am block: %d with tid: %d Result %d \n",blockIdx.x,tid,c[tid]);
    start_index +=1;
    }
}


/*
Matrix Matrix multiplication with a single thread for each row
*/

__global__ void
matrix_matrix_mul_old(int *a, int *b, int *c, int n_row, int n_col, int n_comm)

{
    int tid= threadIdx.x + blockIdx.x * blockDim.x;
    int temp=0;
    while(tid<n_row)
    {
        for (int k=0;k<n_col;k++)
        {
           temp=0; 
            for(int j=0;j<n_comm;j++)
            {
                temp+= a[n_comm*tid+j]* b[j*n_col+k];
            }
            c[tid*n_col+k]=temp;
        }    
            tid+=blockDim.x * gridDim.x;
        
    }
}

/*
  Matrix Matrix multiplication with a single thread for each result element
*/
__global__ void
matrix_matrix_new(int *a, int *b, int *c, int n_row, int n_col, int n_comm)
{
    int tid= threadIdx.x + blockIdx.x *  blockDim.x;
    int temp=0;
    while(tid<n_row*n_col)
    {
        // find the row index of A
        int i=tid / n_col;
        // find the column index of B
        int j=tid % n_col;
        // multiply the row and column
        temp=0;
        for(int k=0;k<n_comm;k++)
        {
         temp+= a[i*n_comm+k]*b[j+k*n_col];
        }
        c[tid]=temp;
        tid+= blockDim.x * gridDim.x;
    }
}


/*
  Matrix Vector multiplication with a block with 4 threads per block, shared block mem and parallel reduce
*/

__global__ void
good_multiplication(int*a,int*b,int *c,int n_col=2,int n_row=2)
{

   __shared__ int intermediate[4];
int tid= threadIdx.x + blockIdx.x * blockDim.x;
int index = (blockDim.x* blockIdx.x)+ threadIdx.x;
int length = blockDim.x;
// Each thread needs two value
intermediate[threadIdx.x]=a[index]*b[threadIdx.x];
printf("\n BlockID:%d,  Tid: %d, index: %d, value:%d \n ",blockIdx.x,tid,threadIdx.x,intermediate[threadIdx.x]);
__syncthreads();



// Now add all the item in intermidate value with parallel reduce. Determine the number of steps required for reduction for each block.
int total_steps=log2f(length);
//printf("Number of step is %d",total_steps);
int active=0;

// start a loop 
while((threadIdx.x<=(length/2))){
printf("%d is active. length = %d\n",threadIdx.x,length);
if(threadIdx.x==(length/2))
{
    //check if the sequence is odd
    
    if(length%2==1){
        printf("%d should copy value\n",threadIdx.x);
        intermediate[threadIdx.x]=intermediate[threadIdx.x+length/2];
    }
    
}
else
{
    intermediate[threadIdx.x]+=intermediate[threadIdx.x+length/2];
}
length = length/2+ length%2;
printf("New length is %d\n",length);
__syncthreads();
//printf("Intermediate sum is %d\n",intermediate[threadIdx.x]);



if(length==1 && threadIdx.x==0 )
{
// write to global memory

    c[blockIdx.x]=intermediate[threadIdx.x];
    printf("\n Thread 0 wrote result");
    break;
}
}
}

void display(int *a,int len)
{
    printf("\n");
    for (int i=0;i<len;i++)
    {
        printf("%d \n",a[i]);
    }
}



/*
  Parallel reduce with elements in intermediate array and result in c array.
*/


while((threadIdx.x<=(length/2))){
printf("%d is active. length = %d\n",threadIdx.x,length);
if(threadIdx.x==(length/2))
{
    //check if the sequence is odd
    
    if(length%2==1){
        printf("%d should copy value\n",threadIdx.x);
        intermediate[threadIdx.x]=intermediate[threadIdx.x+length/2];
    }
    
}
else
{
    intermediate[threadIdx.x]+=intermediate[threadIdx.x+length/2];
}
length = length/2+ length%2;
printf("New length is %d\n",length);
__syncthreads();
//printf("Intermediate sum is %d\n",intermediate[threadIdx.x]);



if(length==1 && threadIdx.x==0 )
{
// write to global memory

    c[blockIdx.x]=intermediate[threadIdx.x];
    printf("\n Thread 0 wrote result");
    break;
}
}


/*
Parallel reduce coalesced
*/
__shared__ int a[5];
117         if(threadIdx.x==0){
118         for (int i=0; i<5;i++){
119         a[i]=i;}}
120         int len=5;
121         while(len/2>0 && threadIdx.x<len)
122         {
123         int required = len/2 + len%2;
124         int offset = blockDim.x; 
125         //printf("\nStart:%d, step:%d,stop:%d\n",start,step,stop);
126         int halfpoint=(len/2);
127         printf("Halfpoint:%d\n ",halfpoint);
128                 for (int i =threadIdx.x; i<required;i+=offset)
129                 {
130                 int temp1=a[i];
131                 int temp2=a[i+halfpoint];
132                 __syncthreads();
133                         
134                 if  (i == (required-1) && i>0)
135                 {
136                 if (required%2==1)
137                         {
138                         //printf("\n copy for %d\n",i);
139                         a[i]=temp2;
140                         }
141                 }
142                 else{a[i]= temp1+temp2;}
143                 __syncthreads();
144                 printf("thread: %d working on:%d and %d,value: %d\n",threadIdx.x,i,i+halfpoint,a[i]);
145                 }
146                 __syncthreads();
147                 len=halfpoint+len%2;
148         }
149         printf("Result: %d  thread%d \n", a[0],threadIdx.x);

